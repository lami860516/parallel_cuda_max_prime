#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <math.h>
#include <stdbool.h>

#define BNUM 180
#define TNUM 1024
unsigned long long MakeNum(bool *number,unsigned long long size){
	unsigned long long i,j,now=0;
	for(i=0;i<size;i++)
		number[i]=0;
	number[2]=1;number[3]=1;
	for(i=5,j=2;i<size;i+=j,j=6-j){
		number[i]=1;
		now++;
	}//printf("%llu %llu\n",now,number[now-1]);
	return size;
    //number[0] = 2;
}

__global__ void running(bool *deviceArr,unsigned long long arrSize){
        int BID=blockIdx.x;       //�϶�����
        int TID=threadIdx.x;      //���������
        //int n=blockDim.x;       //�϶����]�t��������ƥ�
        //int x=BID*n+TID;            //������b�}�C����������m
		//deviceArr[arrSize-1]++;
		unsigned long long i,j,k;
		for(i = BID * TNUM + TID; i < arrSize;i += BNUM * TNUM){
			if(deviceArr[i]==1){
				//for(j=2;i*j<arrSize;j++)
				for (j = 5,k=2; j * i < arrSize;j+=k,k=6-k) 
				{
				  deviceArr[i * j] = 0;
				}
			}
		}
		
		
};
int main(){
	unsigned long long MAXNUM=16000000000;
while(1){
		bool *arr;	
		bool *hostArr;
		bool *deviceArr;
		unsigned long long i,arrSize,temp,biggest;
		float dTime;
		hipEvent_t start,end;
		
		arr = (bool *)malloc(MAXNUM*sizeof(bool));
		hostArr = (bool *)malloc(MAXNUM*sizeof(bool));
		hipEventCreate(&start);
		hipEventCreate(&end);
		arrSize=MakeNum(arr,MAXNUM);
		//printf("%llu %llu\n",arrSize,arr[arrSize-1]);
		/*for(i=0;i<arrSize;i++)
			if(arr[i]==1)
				printf("%llu ",i);
		printf("\n",arr[i]);*/
		hipMalloc((void**) &deviceArr, MAXNUM*sizeof(bool));
		hipMemcpy(deviceArr,arr,sizeof(bool)*MAXNUM,hipMemcpyHostToDevice);
		hipEventRecord(start, 0);
		running<<<BNUM,TNUM>>>(deviceArr,arrSize);
		hipEventRecord(end, 0);
		hipEventSynchronize(end); 
		hipMemcpy(hostArr, deviceArr, MAXNUM*sizeof(bool), hipMemcpyDeviceToHost);
		temp=0;
		for(i=0;i<arrSize;i++){
			if(hostArr[i]==1){
				temp++;
				biggest=i;
				//printf("%llu ",i);
			}
				
		}/*printf("\n");*/
		hipEventElapsedTime(&dTime, start, end);
		printf("2~%llu num:%llu  biggest:%llu  time:%f ms.\n",MAXNUM,temp,biggest,dTime);
		hipFree(deviceArr);
		
		free(arr);free(hostArr);
		MAXNUM+=100000000;
		//if(MAXNUM>=16505000000)break;
		}
        //return 0;
}


