#include<stdio.h>
#include<hip/hip_runtime.h>
#include <string.h>
#include <math.h>
#define MAXNUM 10000000000
#define BNUM 190
#define TNUM 1024
long long MakeNum(int *number,long long size){
	int i,j,now=0;
	for(i=0;i<size;i++)
		number[i]=0;
	number[2]=1;number[3]=1;
	for(i=5,j=2;i<size;i+=j,j=6-j){
		number[i]=1;
		now++;
	}//printf("%llu %llu\n",now,number[now-1]);
	return size;
    //number[0] = 2;
}

__global__ void running(int *deviceArr,long long arrSize){
        int BID=blockIdx.x;       //區塊索引
        int TID=threadIdx.x;      //執行緒索引
        //int n=blockDim.x;       //區塊中包含的執行緒數目
        //int x=BID*n+TID;            //執行緒在陣列中對應的位置
		//deviceArr[arrSize-1]++;
		long long i,j,k;
		for(i = BID * TNUM + TID; i < arrSize;i += BNUM * TNUM){
			if(deviceArr[i]==1){
				for(j=2;i*j<arrSize;j++)
				//for (j = 5,k=2; j * i < arrSize;j+=k,k=6-k) 
				{
				  deviceArr[i * j] = 0;
				}
			}
		}
		
		
};
int main(){
		int *arr;	
		int *hostArr;
		int *deviceArr;
		long long i,j,k,arrSize,temp,biggest;
		float dTime;
		hipEvent_t start,end;
		
		arr = (int *)malloc(MAXNUM*sizeof(int));
		hostArr = (int *)malloc(MAXNUM*sizeof(int));
		hipEventCreate(&start);
		hipEventCreate(&end);
		arrSize=MakeNum(arr,MAXNUM);
		//printf("%llu %llu\n",arrSize,arr[arrSize-1]);
		/*for(i=0;i<arrSize;i++)
			if(arr[i]==1)
				printf("%llu ",i);
		printf("\n",arr[i]);*/
		hipMalloc((void**) &deviceArr, MAXNUM*sizeof(int));
		hipMemcpy(deviceArr,arr,sizeof(int)*MAXNUM,hipMemcpyHostToDevice);
		hipEventRecord(start, 0);
		running<<<BNUM,TNUM>>>(deviceArr,arrSize);
		hipEventRecord(end, 0);
		hipEventSynchronize(end); 
		hipMemcpy(hostArr, deviceArr, MAXNUM*sizeof(int), hipMemcpyDeviceToHost);
		temp=0;
		for(i=0;i<arrSize;i++){
			if(hostArr[i]==1){
				temp++;
				biggest=i;
				//printf("%llu ",i);
			}
				
		}/**/
		hipEventElapsedTime(&dTime, start, end);
		printf("2~%llu num:%llu  biggest:%llu  time:%f.\n",MAXNUM,temp,biggest,dTime);
		hipFree(deviceArr);
		/*
        cudaMalloc((void**) &d, 100*sizeof(Index));
		
        int g=3, b=4, m=g*b;
        running<<<g,b>>>(d);

        cudaMemcpy(h, d, 100*sizeof(Index), cudaMemcpyDeviceToHost);

        for(int i=0; i<m; i++){
            printf("h[%d]={block:%d, thread:%d,%d,%d}\n", i,h[i].block,h[i].thread,h[i].n,h[i].x);
        }

        cudaFree(d);*/
		
        return 0;
}


